#include "cutlass/cutlass.cuh"

#include "cutlass/gemm/device/gemm.h"
#include <mma.h>
#include <vector>
#include <cstdio>
#include <iostream>
#include <cassert>
#include <functional>

hipError_t CutlassSgemmNN(
  int M,
  int N,
  int K,
  float alpha,
  float const *A,
  int lda,
  float const *B,
  int ldb,
  float beta,
  float *C,
  int ldc) {

  
  using RowMajor = cutlass::layout::RowMajor;

  using CutlassGemm = cutlass::gemm::device::Gemm<float,        // Data-type of A matrix
                                                  RowMajor,  // Layout of A matrix
                                                  float,        // Data-type of B matrix
                                                  RowMajor,  // Layout of B matrix
                                                  float,        // Data-type of C matrix
                                                  RowMajor>; // Layout of C matrix

  // Define a CUTLASS GEMM type
  CutlassGemm gemm_operator;

  // Construct the CUTLASS GEMM arguments object.
  //
  // One of CUTLASS's design patterns is to define gemm argument objects that are constructible
  // in host code and passed to kernels by value. These may include pointers, strides, scalars,
  // and other arguments needed by Gemm and its components.
  //
  // The benefits of this pattern are (1.) a structured, composable strategy for passing host-constructible
  // arguments to kernels and (2.) minimized initialization overhead on kernel entry.
  //
  CutlassGemm::Arguments args({M , N, K},  // Gemm Problem dimensions
                              {A, lda},    // Tensor-ref for source matrix A
                              {B, ldb},    // Tensor-ref for source matrix B
                              {C, ldc},    // Tensor-ref for source matrix C
                              {C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                              {alpha, beta}); // Scalars used in the Epilogue

  //
  // Launch the CUTLASS GEMM kernel.
  //
  
  cutlass::Status status = gemm_operator(args);

  //
  // Return a hipError_t if the CUTLASS GEMM operator returned an error code.
  //

  if (status != cutlass::Status::kSuccess) {
    printf("cutlass error\n");
    return hipErrorUnknown;
  }

  // Return success, if no errors were encountered.
  return hipSuccess;
}




template <typename T>
void cutlass_gemm(T *dA, T *dB, T*dC, int m, int n, int k) {
    CutlassSgemmNN(m ,n, k, 1.0, dA, k, dB, n, 0.0, dC, n);
}

template void cutlass_gemm<float>(float *dA, float *dB, float *dC, int m, int n, int k);
